#include "hip/hip_runtime.h"
/* Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This is a simple test showing performance and usability
 * improvements with large kernel parameters introduced in CUDA 12.1
 */
#include <chrono>
#include <iostream>
#include <cassert>
#include <vector>
#include <algorithm>

// Utility includes
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace std::chrono;

//#define TEST_ITERATIONS     (1000)
//#define TOTAL_PARAMS        (8188)  // ints
//#define KERNEL_PARAM_LIMIT  (8188)  // ints
//#define CONSTANT_PARAM_LIMIT (16384)  // ints
//#define CONST_COPIED_PARAMS (CONSTANT_PARAM_LIMIT - 0)

#define TEST_ITERATIONS     (1)
#define KERNEL_PARAM_LIMIT  (7680)  // ints //5460
#define KERNEL_PARAM_READ   (7680)
#define CONSTANT_PARAM_LIMIT (16384)  // ints //4680
#define CONSTANT_PARAM_READ (0)  // ints //4680
#define CONST_STRIDE (8)
#define CONST_COPIED_PARAMS (CONSTANT_PARAM_LIMIT)
#define STRIDE (4)
#define NUMWAVES (1)
#define WARPSIZE (32)
#define MEMDATA (16777216)

typedef int IDT;

__constant__ int excess_params[CONST_COPIED_PARAMS];

#define SM_SIZE 0x1
//__shared__ int sharedData[SM_SIZE];

typedef struct {
  IDT param[KERNEL_PARAM_LIMIT];
} param_large_t;


// Kernel with 32,764 byte kernel parameter limit
__global__ void kernelLargeParam(__grid_constant__ const param_large_t p, int *a, int *result) {
  int tid = (threadIdx.x & 0x1F);
  int wid = (threadIdx.x >> 5);
  IDT tmp = 0;
  int num_trans = (CONSTANT_PARAM_READ+ KERNEL_PARAM_READ) / CONST_STRIDE;


  if (tid == 0 ) {
#pragma unroll
    for (int i = 0; i < KERNEL_PARAM_LIMIT; i += 64) {
      tmp += p.param[i];
    }
#pragma unroll
    for (int i = 0; i < CONSTANT_PARAM_LIMIT; i += 16) {
      tmp += excess_params[i];
    }
  }
  __syncthreads();

  for (int i = 0; i < MEMDATA; ++i) {
    tmp += a[i];
  }

  volatile int start_time, end_time;
  if (tid == 0) {
    start_time = clock();

#pragma unroll
    //for (int i = KERNEL_PARAM_LIMIT - KERNEL_PARAM_READ; i < KERNEL_PARAM_LIMIT; i += CONST_STRIDE) {
    for (int i = 0; i < KERNEL_PARAM_LIMIT; i += CONST_STRIDE) {
      tmp += p.param[i];
    }
#pragma unroll
    for (int i = 0; i < CONSTANT_PARAM_LIMIT; i += CONST_STRIDE) {
      //tmp += excess_params[i];
    }

    end_time = clock();
    float p = float(end_time - start_time)/num_trans;
    printf("[%d]: %d, %d, %f\n", end_time - start_time, start_time, end_time, p);
  }
  if (tid == 0) {
    result[wid * STRIDE] = tmp;
    result[wid*STRIDE + 1] = start_time;
    result[wid*STRIDE + 2] = end_time;
  }

}

/*
static void report_time(std::chrono::time_point<std::chrono::steady_clock> start,
                        std::chrono::time_point<std::chrono::steady_clock> end,
                        int iters) {
  auto usecs = duration_cast<duration<float,
                                      microseconds::period>>(end - start);
  cout << usecs.count() / iters << endl;
}
*/

int main() {
  int rc;
  hipFree(0);

  param_large_t p_large;

  // storage for computed result
  constexpr int WAVESIZE = NUMWAVES*WARPSIZE;
  constexpr int NOUT = NUMWAVES * STRIDE;

  std::vector<int> ha(MEMDATA);
  for (int i = 0; i < MEMDATA; ++i) {
    ha[i] = i;
  }

  int* d_a;
  checkCudaErrors(hipMalloc(&d_a, MEMDATA*sizeof(int)));

  int *d_result;
  int h_result[NOUT];
  checkCudaErrors(hipMalloc(&d_result, NOUT*sizeof(int)));

  int expected_result = 0;

  // fill in data for validation
  for (int i = 0; i < KERNEL_PARAM_LIMIT; ++i) {
    p_large.param[i] = (i & 0xFF);
    expected_result += (i & 0xFF);
  }

  checkCudaErrors(hipMemcpy(d_a, &ha[0], MEMDATA * sizeof(int), hipMemcpyHostToDevice));
  kernelLargeParam<<<1,WAVESIZE>>>(p_large, d_a, d_result);
  checkCudaErrors(hipMemcpy(&h_result, d_result, NOUT*sizeof(int), hipMemcpyDeviceToHost));
  checkCudaErrors(hipDeviceSynchronize());

  for (int i = 0; i < NUMWAVES; ++i) {
    int offset = i * STRIDE;
    printf("W:[%d] Large Executed Cycles: [%d - %d], [%d]\n", i, h_result[offset + 1], h_result[offset + 2], h_result[offset + 2] - h_result[offset + 1]);
  }

  if(h_result[0] != expected_result) {
    std::cout << "Test failed" << std::endl;
	 rc=-1;
	 goto Exit;    
  }

  rc=0;
Exit:
  // cleanup
  hipFree(d_result);
  return rc;
}
